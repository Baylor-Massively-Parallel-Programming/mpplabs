
#include <hip/hip_runtime.h>
/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

#define BLOCK_SIZE 512
#define SIMPLE

__global__ void reduction(float *out, float *in, unsigned size)
{
    /********************************************************************
    Load a segment of the input vector into shared memory
    Traverse the reduction tree
    Write the computed sum to the output vector at the correct index
    ********************************************************************/

    // INSERT KERNEL CODE HERE

#ifdef SIMPLE
    __shared__ float in_s[2*BLOCK_SIZE];
    int idx = 2 * blockIdx.x * blockDim.x + threadIdx.x;

    in_s[threadIdx.x]            = ((idx              < size)? in[idx]:            0.0f);
    in_s[threadIdx.x+BLOCK_SIZE] = ((idx + BLOCK_SIZE < size)? in[idx+BLOCK_SIZE]: 0.0f);

    




#else
    __shared__ float in_s[BLOCK_SIZE];
    int idx = 2 * blockIdx.x * blockDim.x + threadIdx.x;

    in_s[threadIdx.x] = ((idx              < size)? in[idx]:            0.0f) +
		 	((idx + BLOCK_SIZE < size)? in[idx+BLOCK_SIZE]: 0.0f);

    


#endif

    

}
